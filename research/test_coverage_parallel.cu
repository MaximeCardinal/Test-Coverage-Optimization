#include "hip/hip_runtime.h"
#define THREADS_PER_BLOCK 1024
#define NB_BLOCKS 1

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

// Runs the coverage program
__global__ void coverage(char** input, int iterations)
{
    int thread_index = threadIdx.x;
    int nbr_threads = blockDim.x;
    for (int i = thread_index; i < iterations; i = i + nbr_threads) {
        system(input[i]);
        system(input[i + 1]);
    }
}

int main(int argc, char* argv[])
{
    // Check for user input validity
    if (argc != 3) {
        printf("Wrong number of arguments\n");
        return 1;
    }

    // Get user input
    char* input_filename = argv[1];
    int file_length = atoi(argv[2]);

    // Allocate memory
    char** input;
    hipMallocManaged((void**) &input, file_length * sizeof(char*));
    for (int i = 0; i < file_length; i++)
        hipMallocManaged((void**)&input[i], 1024 * sizeof(char));

    // Parse input file
    //char line[1024];
    FILE* input_file = fopen(input_filename, "r");

    for (int i = 0; i < file_length; i++) {
        fgets(input[i], 1024, input_file);
    }
    fclose(input_file);

    coverage << <NB_BLOCKS, THREADS_PER_BLOCK >> > (input, file_length/2);
    hipDeviceSynchronize();

    //Clean
    hipFree(input);
    return 0;
}